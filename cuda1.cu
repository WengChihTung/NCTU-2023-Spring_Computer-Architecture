#include "hip/hip_runtime.h"
#include "parameters.h"


__global__ void cuda_kernel(int **dB, int **dA, IndexSave **dInd)
{	
	// int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// if(idx < SIZE * SIZE){
	// 	int m = idx / SIZE;
	// 	int n = idx % SIZE;
	//  	dInd[m][n].blockInd_x = blockIdx.x;
	//  	dInd[m][n].threadInd_x = threadIdx.x;
	//  	dInd[m][n].head = idx;
	//  	dInd[m][n].stripe = 1;
	//  	dB[m][n] += dA[m][n];
	//  	dB[m][n] += dA[m][n];

	// }
	int i = 0;
	int TotalThread = blockDim.x * gridDim.x;
	int      stripe = SIZE * SIZE / TotalThread;
	int        head = (blockIdx.x * blockDim.x + threadIdx.x) * stripe;
	int     LoopLim = head + stripe;
	
	for(i = head; i < LoopLim; i++){
		int m = i / SIZE;
		int n = i % SIZE;
		dInd[m][n].blockInd_x = blockIdx.x;
		dInd[m][n].threadInd_x = threadIdx.x;
		dInd[m][n].head = head;
		dInd[m][n].stripe = stripe;
		dB[m][n] += dA[m][n];
		dB[m][n] += dA[m][n];
	}
};


float GPU_kernel(int **B, int **A, IndexSave **indsave){

	int** dA; 
	int** dB;
	IndexSave** dInd;

	// Creat Timing Event
  	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop); 	

	// Allocate Memory Space on Device
	hipMalloc((void**)&dA, sizeof(int*) * SIZE);
	hipMalloc((void**)&dB, sizeof(int*) * SIZE);

	// Allocate Memory Space on Device (for observation)
	hipMalloc((void**)&dInd, sizeof(IndexSave*) * SIZE);
	
	for(int i = 0; i < SIZE; i++){
		hipMalloc((void**)&dA[i], sizeof(int) * SIZE);
		hipMalloc((void**)&dB[i], sizeof(int) * SIZE);
		hipMalloc((void**)&dInd[i], sizeof(IndexSave) * SIZE);
	}

	// Copy Data to be Calculated
	for(int i = 0; i < SIZE; i++){
		hipMemcpy(dA[i], A[i], sizeof(int)*SIZE, hipMemcpyHostToDevice);
		hipMemcpy(dB[i], B[i], sizeof(int)*SIZE, hipMemcpyHostToDevice);
		// Copy Data (indsave array) to device
		hipMemcpy(dInd[i], indsave[i], sizeof(IndexSave)*SIZE, hipMemcpyHostToDevice);
	}

	
	// Start Timer
	hipEventRecord(start, 0);

	// Launch Kernel
	dim3 dimGrid(4);
	dim3 dimBlock(4);
	cuda_kernel<<<dimGrid,dimBlock>>>(dB,dA,dInd);
	//hipDeviceSynchronize();

	// Stop Timer
	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop); 

	// Copy Output back
	for(int i = 0; i < SIZE; i++){
		hipMemcpy(A[i], dA[i], sizeof(int)*SIZE, hipMemcpyDeviceToHost);
		hipMemcpy(B[i], dB[i], sizeof(int)*SIZE, hipMemcpyDeviceToHost);
		
		hipMemcpy(indsave[i], dInd[i], sizeof(IndexSave)*SIZE, hipMemcpyDeviceToHost);
	}

	// Release Memory Space on Device
	for(int i = 0; i < SIZE; i++){
		hipFree(dA[i]);
		hipFree(dB[i]);
		hipFree(dInd[i]);
	}
	hipFree(dA);
	hipFree(dB);
	hipFree(dInd);

	// Calculate Elapsed Time
  	float elapsedTime; 
  	hipEventElapsedTime(&elapsedTime, start, stop);  

	return elapsedTime;
}
