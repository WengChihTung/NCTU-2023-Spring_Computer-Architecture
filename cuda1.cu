#include "hip/hip_runtime.h"
#include "parameters.h"


__global__ void cuda_kernel(int dB[][SIZE],int dA[][SIZE],IndexSave dInd[][SIZE])
{	
	int i=0;
	int TotalThread = blockDim.x*gridDim.x;
	int stripe = SIZE * SIZE / TotalThread;
	int head   = (blockIdx.x*blockDim.x + threadIdx.x)*stripe;
	int LoopLim = head+stripe;
	
	for(i=head ; i<LoopLim ; i++ ){
		for(int j=0; j<SIZE; j++){
			dInd[i][j].blockInd_x = blockIdx.x;
			dInd[i][j].threadInd_x = threadIdx.x;
			dInd[i][j].head = head;
			dInd[i][j].stripe = stripe;
			dB[i][j] += dA[i][j];
			dB[i][j] += dA[i][j];
		}
	}
};


float GPU_kernel(int B[][SIZE],int A[][SIZE],IndexSave indsave[][SIZE]){

	//int dA[SIZE][SIZE],dB[SIZE][SIZE];
	//IndexSave dInd[SIZE][SIZE];

	// Creat Timing Event
  	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop); 	

	// Allocate Memory Space on Device

	// Allocate Memory Space on Device (for observation)
	//hipMalloc((void**)&dInd,sizeof(IndexSave)*SIZE);

	// Copy Data to be Calculated
	//hipMemcpy(dA, A, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);
	//hipMemcpy(dB, B, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);

	// Copy Data (indsave array) to device
	//hipMemcpy(dInd, indsave, sizeof(IndexSave)*SIZE*SIZE, hipMemcpyHostToDevice);
	
	// Start Timer
	hipEventRecord(start, 0);

	// Lunch Kernel
	dim3 dimGrid(2);
	dim3 dimBlock(4);
	cuda_kernel<<<dimGrid,dimBlock>>>(B,A,indsave);

	// Stop Timer
	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop); 

	// Copy Output back
	//hipMemcpy(A, dA, sizeof(int)*SIZE*SIZE, hipMemcpyDeviceToHost);
	//hipMemcpy(B, dB, sizeof(int)*SIZE*SIZE, hipMemcpyDeviceToHost);
	
	//hipMemcpy(indsave, dInd, sizeof(IndexSave)*SIZE*SIZE, hipMemcpyDeviceToHost);

	// Release Memory Space on Device
	//hipFree(dA);
	//hipFree(dB);
	//hipFree(dInd);

	// Calculate Elapsed Time
  	float elapsedTime; 
  	hipEventElapsedTime(&elapsedTime, start, stop);  

	return elapsedTime;
}
