#include "hip/hip_runtime.h"
#include "parameters.h"


__global__ void cuda_kernel(int **dB, int **dA, IndexSave **dInd)
{
	int i = 0;
	int  stripe = blockDim.x * gridDim.x;
	int    head = blockIdx.x * blockDim.x + threadIdx.x;

	for(i = head; i < SIZE * SIZE; i += stripe){
		int m = i / SIZE;
		int n = i % SIZE;
		dInd[m][n].blockInd_x = blockIdx.x;
		dInd[m][n].threadInd_x = threadIdx.x;
		dInd[m][n].head = head;
		dInd[m][n].stripe = stripe;
		//Brightness Increase
		dB[m][n] += dA[m][n];
		dB[m][n] += dA[m][n];
		//Thresholding
		if(dB[m][n] > 127)
			dB[m][n] = 255;
		else
			dB[m][n] = 0;
	}
};


float GPU_kernel(int **B, int **A, IndexSave **indsave){

	int** dA;
	int** dB;
	IndexSave** dInd;

	// Creat Timing Event
  	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop);

	// Allocate Memory Space on Device
	hipMalloc((void**)&dA, sizeof(int*) * SIZE);
	hipMalloc((void**)&dB, sizeof(int*) * SIZE);

	// Allocate Memory Space on Device (for observation)
	hipMalloc((void**)&dInd, sizeof(IndexSave*) * SIZE);

	for(int i = 0; i < SIZE; i++){
		hipMalloc((void**)&dA[i], sizeof(int) * SIZE);
		hipMalloc((void**)&dB[i], sizeof(int) * SIZE);
		hipMalloc((void**)&dInd[i], sizeof(IndexSave) * SIZE);
	}

	// Copy Data to be Calculated
	for(int i = 0; i < SIZE; i++){
		hipMemcpy(dA[i], A[i], sizeof(int)*SIZE, hipMemcpyHostToDevice);
		hipMemcpy(dB[i], B[i], sizeof(int)*SIZE, hipMemcpyHostToDevice);
		// Copy Data (indsave array) to device
		hipMemcpy(dInd[i], indsave[i], sizeof(IndexSave)*SIZE, hipMemcpyHostToDevice);
	}


	// Start Timer
	hipEventRecord(start, 0);

	// Launch Kernel
	dim3 dimGrid(4);
	dim3 dimBlock(4);
	cuda_kernel<<<dimGrid,dimBlock>>>(dB,dA,dInd);
	//hipDeviceSynchronize();

	// Stop Timer
	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop);

	// Copy Output back
	for(int i = 0; i < SIZE; i++){
		hipMemcpy(A[i], dA[i], sizeof(int)*SIZE, hipMemcpyDeviceToHost);
		hipMemcpy(B[i], dB[i], sizeof(int)*SIZE, hipMemcpyDeviceToHost);

		hipMemcpy(indsave[i], dInd[i], sizeof(IndexSave)*SIZE, hipMemcpyDeviceToHost);
	}

	// Release Memory Space on Device
	for(int i = 0; i < SIZE; i++){
		hipFree(dA[i]);
		hipFree(dB[i]);
		hipFree(dInd[i]);
	}
	hipFree(dA);
	hipFree(dB);
	hipFree(dInd);

	// Calculate Elapsed Time
  	float elapsedTime;
  	hipEventElapsedTime(&elapsedTime, start, stop);

	return elapsedTime;
}
